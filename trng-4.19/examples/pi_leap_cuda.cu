#include "hip/hip_runtime.h"
// Copyright (c) 2000-2010, Heiko Bauke
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
// 
//   * Redistributions of source code must retain the above copyright
//     notice, this list of conditions and the following disclaimer.  
// 
//   * Redistributions in binary form must reproduce the above
//     copyright notice, this list of conditions and the following
//     disclaimer in the documentation and/or other materials provided
//     with the distribution.  
// 
//   * Neither the name of the copyright holder nor the names of its
//     contributors may be used to endorse or promote products derived
//     from this software without specific prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
// "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
// LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
// FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
// COPYRIGHT HOLDERS OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
// INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
// HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
// STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED
// OF THE POSSIBILITY OF SUCH DAMAGE.

#include <cstdlib>
#include <iostream>
#include <trng/yarn2.hpp>
#include <trng/uniform01_dist.hpp>

__global__
void parallel_pi(long samples, trng::yarn2 *rx, trng::yarn2 *ry, long *in) {
  long rank=threadIdx.x;
  long size=blockDim.x;
  trng::uniform01_dist<float> u;        // random number distribution
  in[rank]=0;                           // local number of points in circle
  for (long i=rank*samples/size; i<(rank+1)*samples/size; ++i) {
    float x=u(rx[rank]), y=u(ry[rank]); // choose random x- and y-coordinates
    if (x*x+y*y<=1)                     // is point in circle?
      ++in[rank];                       // increase thread-local counter
  }
}

int main(int argc, char *argv[]) {
  const long samples=1000000l;             // total number of points in square
  const int size=128;                      // number of threads
  trng::yarn2 *rx=new trng::yarn2[size];   // random number engines
  trng::yarn2 *ry=new trng::yarn2[size];   // random number engines
  for (int rank=0; rank<size; ++rank) {
    rx[rank].split(2, 0);                  // choose sub-stream no. 0 out of 2 streams
    ry[rank].split(2, 1);                  // choose sub-stream no. 1 out of 2 streams
    rx[rank].split(size, rank);            // choose sub-stream no. rank out of size streams
    ry[rank].split(size, rank);            // choose sub-stream no. rank out of size streams
  }
  // copy random number engines to CUDA device
    trng::yarn2 *rx_device, *ry_device;
  hipMalloc(&rx_device, size*sizeof(*rx_device));
  hipMalloc(&ry_device, size*sizeof(*ry_device));
  hipMemcpy(rx_device, rx, size*sizeof(*rx), hipMemcpyHostToDevice);
  hipMemcpy(ry_device, ry, size*sizeof(*ry), hipMemcpyHostToDevice);
  // memory for thread local results
  long *in_device;
  hipMalloc(&in_device, size*sizeof(*in_device));
  // start parallel Monte Carlo
  parallel_pi<<<1, size>>>(samples, rx_device, ry_device, in_device);
  // gather results
  long *in=new long[size];
  hipMemcpy(in, in_device, size*sizeof(*in), hipMemcpyDeviceToHost);
  long sum=0;
  for (int rank=0; rank<size; ++rank)
    sum+=in[rank];
  // print result
  std::cout << "pi = " << 4.0*sum/samples << std::endl;
  return EXIT_SUCCESS;
}
