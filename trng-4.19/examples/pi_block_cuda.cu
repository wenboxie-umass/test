#include "hip/hip_runtime.h"
// Copyright (c) 2000-2010, Heiko Bauke
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
// 
//   * Redistributions of source code must retain the above copyright
//     notice, this list of conditions and the following disclaimer.  
// 
//   * Redistributions in binary form must reproduce the above
//     copyright notice, this list of conditions and the following
//     disclaimer in the documentation and/or other materials provided
//     with the distribution.  
// 
//   * Neither the name of the copyright holder nor the names of its
//     contributors may be used to endorse or promote products derived
//     from this software without specific prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
// "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
// LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
// FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
// COPYRIGHT HOLDERS OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
// INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
// HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
// STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED
// OF THE POSSIBILITY OF SUCH DAMAGE.

#include <cstdlib>
#include <iostream>
#include <trng/yarn5s.hpp>
#include <trng/uniform01_dist.hpp>

__global__
void parallel_pi(long samples, long *in, trng::yarn5s r) {
  long rank=threadIdx.x;
  long size=blockDim.x;
  r.jump(2*(rank*samples/size));      // jump ahead
  trng::uniform01_dist<float> u;      // random number distribution
  in[rank]=0;                         // local number of points in circle
  for (long i=rank*samples/size; i<(rank+1)*samples/size; ++i) {
    float x=u(r), y=u(r);             // choose random x- and y-coordinates
    if (x*x+y*y<=1)                   // is point in circle?
      ++in[rank];                     // increase thread-local counter
  }
}

int main(int argc, char *argv[]) {
  const long samples=1000000l;             // total number of points in square
  const int size=128;                      // number of threads
  long *in_device;
  hipMalloc(&in_device, size*sizeof(*in_device));
  trng::yarn5s r;
  // start parallel Monte Carlo
  parallel_pi<<<1, size>>>(samples, in_device, r);
  // gather results
  long *in=new long[size];
  hipMemcpy(in, in_device, size*sizeof(*in), hipMemcpyDeviceToHost);
  long sum=0;
  for (int rank=0; rank<size; ++rank) 
    sum+=in[rank];
  // print result
  std::cout << "pi = " << 4.0*sum/samples << std::endl;
  return EXIT_SUCCESS;
}
